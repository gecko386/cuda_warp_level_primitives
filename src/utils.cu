/*
 * utils.cu
 *
 * Author: Ruben
 */

#include <iostream>
#include <hip/hip_runtime_api.h>
#include "utils.h"

KernelTimer::KernelTimer()
{
	cudaCheckError(hipDeviceSynchronize());
	start = std::chrono::steady_clock::now();
}

KernelTimer::~KernelTimer()
{
	cudaCheckError(hipDeviceSynchronize());
	auto end = std::chrono::steady_clock::now();
	auto elapsed =
			std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
	std::cout << "kernel ran in " << elapsed << " ms\n";
}
